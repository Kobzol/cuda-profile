#include "hip/hip_runtime.h"
#include <CuprRuntime.h>
#include <hip/hip_runtime_api.h>

__global__ void bankConflict(int *p, int *q)
{
    __shared__ float mem[64];

    int index = 0;
    if (threadIdx.x < 16)
    {
        index = threadIdx.x;
    }
    else if (threadIdx.x < 20)
    {
        index = threadIdx.x + 16;
    }
    else index = threadIdx.x;

    mem[index] = threadIdx.x;
}

int main(int argc, char** argv)
{
    const int COUNT = 32;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    bankConflict<<<1, COUNT>>>(dPtr, dPtr);

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);

    return 0;
}
