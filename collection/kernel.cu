#include "hip/hip_runtime.h"
#include "general.h"
#include "device/CuprRuntime.h"

__global__ void kernel(int *p)
{
    *p = 5;
}

void cudaTest()
{
    const int COUNT = 32;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    kernel<<<32, COUNT>>>(dPtr);

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);
}
