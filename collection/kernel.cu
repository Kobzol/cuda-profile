#include "hip/hip_runtime.h"
#include "general.h"
#include "device/CuprRuntime.h"

__global__ void kernel(int *p)
{
    p[threadIdx.x] = 5;
}

void cudaTest()
{
    const int COUNT = 16;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    kernel<<<2, COUNT>>>(dPtr);

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);
}
