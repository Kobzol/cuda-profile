#include "hip/hip_runtime.h"
#include "general.h"
#include "device/Runtime.h"

__global__ void kernel(int *p)
{
    *p = 5;
}

void cudaTest()
{
    const int COUNT = 256;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    kernel<<<256, COUNT>>>(dPtr);

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);
}
