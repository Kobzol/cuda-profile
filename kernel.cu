#include "hip/hip_runtime.h"
#include "general.h"
#include "runtime/Runtime.h"

__global__ void kernel(int* p)
{
    int a = *p;
}

void cudaTest()
{
    const int COUNT = 1;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    kernel<<<1, COUNT>>>(dPtr);

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);
}
