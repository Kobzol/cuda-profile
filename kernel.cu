#include "hip/hip_runtime.h"
#include "general.h"
#include "runtime/Runtime.h"

__global__ void kernel(int* p)
{
    p[threadIdx.x] = threadIdx.x;
}

void cudaTest()
{
    const int COUNT = 64;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    kernel<<<1, COUNT>>>(dPtr);

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);
}
