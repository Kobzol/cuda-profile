#include "hip/hip_runtime.h"
#include "general.h"
#include "runtime/Runtime.h"

__global__ void kernel(int* p)
{
    p[threadIdx.x] = threadIdx.x;
}

void cudaTest()
{
    __cu_initMemory();

    const int COUNT = 10;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    __cu_kernelStart();
    kernel<<<1, COUNT>>>(dPtr);
    __cu_kernelEnd();

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);
}
