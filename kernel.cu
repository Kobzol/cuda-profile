#include "hip/hip_runtime.h"
#include "general.h"
#include "runtime/Runtime.h"

__constant__ int constArr[10] = { 0, 1 };

__global__ void kernel()
{
    __shared__ int arr[32];
    arr[5] = constArr[1];
}

void cudaTest()
{
    const int COUNT = 1;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    kernel<<<1, COUNT>>>();

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);
}
