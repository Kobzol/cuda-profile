#include "hip/hip_runtime.h"
#include "general.h"
#include "runtime/Runtime.h"

__global__ void kernel()
{
    __shared__ int arr;
    printf("%p\n", &arr);
}

void cudaTest()
{
    const int COUNT = 2;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    kernel<<<1, 1>>>();

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);
}
