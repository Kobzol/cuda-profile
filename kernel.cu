#include "hip/hip_runtime.h"
#include "general.h"
#include "runtime/Runtime.h"

__global__ void kernel(int *p)
{
    *p = 5;
    *p = 5;
    *p = 5;
}

void cudaTest()
{
    const int COUNT = 1;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    kernel<<<1, COUNT>>>(dPtr);

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);
}
