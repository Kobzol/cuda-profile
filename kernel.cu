#include "hip/hip_runtime.h"
#include "general.h"
#include "runtime/Runtime.h"

__global__ void kernel(int *p)
{
    for (int i = 0; i < 10; i++)
    {
        p[i] = p[i + 1];
    }
}

void cudaTest()
{
    const int COUNT = 64;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    kernel<<<32, COUNT>>>(dPtr);

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);
}
