#include "hip/hip_runtime.h"
#include "general.h"
#include "runtime/Runtime.h"

__global__ void kernel(int* p)
{
    *p = threadIdx.x;
}

void cudaTest()
{
    const int COUNT = 10;

    int* dPtr;
    int data[COUNT] = { 0 };

    hipMalloc((void**) &dPtr, sizeof(data));
    hipMemcpy(dPtr, data, sizeof(data), hipMemcpyHostToDevice);

    printf("HOST: %p\n", dPtr);

    kernel<<<1, COUNT>>>(dPtr);

    int ptr[COUNT];
    hipMemcpy(ptr, dPtr, sizeof(data), hipMemcpyDeviceToHost);

    hipFree(dPtr);
}
